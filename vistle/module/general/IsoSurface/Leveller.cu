#include "hip/hip_runtime.h"
﻿//
//This code is used for both IsoCut and IsoSurface!
//

#include <sstream>
#include <iomanip>
#include <core/index.h>
#include <core/scalar.h>
#include <core/unstr.h>
#include <core/triangles.h>
#include <core/shm.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include "tables.h"
#include "Leveller.h"

using namespace vistle;


const int MaxNumData = 6;


struct HostData {

   Scalar m_isovalue;
   int m_numInVertData, m_numInVertDataI;
   int m_numInCellData, m_numInCellDataI;
   IsoDataFunctor m_isoFunc;
   const Index *m_el;
   const Index *m_cl;
   const unsigned char *m_tl;
   std::vector<Index> m_caseNums;
   std::vector<Index> m_numVertices;
   std::vector<Index> m_LocationList;
   std::vector<Index> m_ValidCellVector;
   const Scalar *m_x;
   const Scalar *m_y;
   const Scalar *m_z;
   std::vector<vistle::shm_ref<vistle::shm_array<Scalar, shm<Scalar>::allocator>>> m_outVertData, m_outCellData;
   std::vector<vistle::shm_ref<vistle::shm_array<Index, shm<Index>::allocator>>> m_outVertDataI, m_outCellDataI;
   std::vector<const Scalar*> m_inVertPtr, m_inCellPtr;
   std::vector<const Index*> m_inVertPtrI, m_inCellPtrI;
   std::vector<Scalar*> m_outVertPtr, m_outCellPtr;
   std::vector<Index *> m_outVertPtrI, m_outCellPtrI;

   typedef const Index *IndexIterator;
   typedef std::vector<Index>::iterator VectorIndexIterator;

   HostData(Scalar isoValue
            , IsoDataFunctor isoFunc
            , const Index *el
            , const unsigned char *tl
            , const Index *cl
            , const Scalar *x
            , const Scalar *y
            , const Scalar *z
            )
      : m_isovalue(isoValue)
      , m_numInVertData(0)
      , m_numInVertDataI(0)
      , m_numInCellData(0)
      , m_numInCellDataI(0)
      , m_isoFunc(isoFunc)
      , m_el(el)
      , m_cl(cl)
      , m_tl(tl)
      , m_x(x)
      , m_y(y)
      , m_z(z)
   {
      m_inVertPtr.push_back(&x[0]);
      m_inVertPtr.push_back(&y[0]);
      m_inVertPtr.push_back(&z[0]);

      for(size_t i = 0; i < m_inVertPtr.size(); i++){
         m_outVertData.emplace_back(vistle::ShmVector<Scalar>::create(0));
         m_outVertPtr.push_back(NULL);
      }
      m_numInVertData = m_inVertPtr.size();
   }

   void addmappeddata(const Scalar *mapdata){

      m_inVertPtr.push_back(mapdata);
      m_outVertData.push_back(vistle::ShmVector<Scalar>::create(0));
      m_outVertPtr.push_back(NULL);
      m_numInVertData = m_inVertPtr.size();
   }

   void addmappeddata(const Index *mapdata){

      m_inVertPtrI.push_back(mapdata);
      m_outVertDataI.push_back(vistle::ShmVector<Index>::create(0));
      m_outVertPtrI.push_back(NULL);
      m_numInVertDataI = m_inVertPtrI.size();
   }

   void addcelldata(const Scalar *mapdata){

      m_inCellPtr.push_back(mapdata);
      m_outCellData.push_back(vistle::ShmVector<Scalar>::create(0));
      m_outCellPtr.push_back(NULL);
      m_numInCellData = m_inCellPtr.size();
   }

   void addcelldata(const Index *mapdata){

      m_inCellPtrI.push_back(mapdata);
      m_outCellDataI.push_back(vistle::ShmVector<Index>::create(0));
      m_outCellPtrI.push_back(NULL);
      m_numInCellDataI = m_inCellPtrI.size();
   }
};

struct DeviceData {

   Scalar m_isovalue;
   int m_numInVertData, m_numInVertDataI;
   int m_numInCellData, m_numInCellDataI;
   IsoDataFunctor m_isoFunc;
   thrust::device_vector<Index> m_el;
   thrust::device_vector<Index> m_cl;
   thrust::device_vector<unsigned char> m_tl;
   thrust::device_vector<Index> m_caseNums;
   thrust::device_vector<Index> m_numVertices;
   thrust::device_vector<Index> m_LocationList;
   thrust::device_vector<Index> m_ValidCellVector;
   thrust::device_vector<Scalar> m_x;
   thrust::device_vector<Scalar> m_y;
   thrust::device_vector<Scalar> m_z;
   std::vector<thrust::device_vector<Scalar> *> m_outVertData, m_outCellData;
   std::vector<thrust::device_vector<Index> *> m_outVertDataI, m_outCellDataI;
   std::vector<thrust::device_ptr<Scalar> > m_inVertPtr, m_inCellPtr;
   std::vector<thrust::device_ptr<Index> > m_inVertPtrI, m_inCellPtrI;
   std::vector<thrust::device_ptr<Scalar> > m_outVertPtr, m_outCellPtr;
   std::vector<thrust::device_ptr<Index> > m_outVertPtrI, m_outCellPtrI;
   typedef const Index *IndexIterator;
   //typedef thrust::device_vector<Index>::iterator IndexIterator;

   DeviceData(Scalar isoValue
              , IsoDataFunctor isoFunc
              , Index nelem
              , const Index *el
              , const unsigned char *tl
              , Index nconn
              , const Index *cl
              , Index ncoord
              , const Scalar *x
              , const Scalar *y
              , const Scalar *z)
   : m_isovalue(isoValue)
   , m_isoFunc(isoFunc)
   , m_el(el, el+nelem)
   , m_cl(cl, cl+nconn)
   , m_tl(tl, tl+nelem)
   , m_x(x, x+ncoord)
   , m_y(y, y+ncoord)
   , m_z(z, z+ncoord)
   {
      m_inVertPtr.push_back(m_x.data());
      m_inVertPtr.push_back(m_y.data());
      m_inVertPtr.push_back(m_z.data());

      for(size_t i = 0; i < m_inVertPtr.size(); i++){
         m_outVertData.push_back(new thrust::device_vector<Scalar>);
      }
      m_outVertPtr.resize(m_inVertPtr.size());
      m_numInVertData = m_inVertPtr.size();
      for(size_t i = 0; i < m_inVertPtrI.size(); i++){
         m_outVertDataI.push_back(new thrust::device_vector<Index>);
      }
      m_outVertPtrI.resize(m_inVertPtrI.size());
      m_numInVertDataI = m_inVertPtrI.size();
   }
};

template<class Data>
struct process_Cell {
   process_Cell(Data &data) : m_data(data) {
      for (int i = 0; i < m_data.m_numInVertData; i++){
         m_data.m_outVertPtr[i] = m_data.m_outVertData[i]->data();
      }
      for (int i = 0; i < m_data.m_numInVertDataI; i++){
         m_data.m_outVertPtrI[i] = m_data.m_outVertDataI[i]->data();
      }
      for (int i = 0; i < m_data.m_numInCellData; i++){
         m_data.m_outCellPtr[i] = m_data.m_outCellData[i]->data();
      }
      for (int i = 0; i < m_data.m_numInCellDataI; i++){
         m_data.m_outCellPtrI[i] = m_data.m_outCellDataI[i]->data();
      }
   }

   Data &m_data;

   __host__ __device__
   void operator()(Index ValidCellIndex) {

      const Index CellNr = m_data.m_ValidCellVector[ValidCellIndex];
      const Index Cellbegin = m_data.m_el[CellNr];
      const Index Cellend = m_data.m_el[CellNr+1];
      const Index numVert = m_data.m_numVertices[ValidCellIndex];
      const auto &cl = &m_data.m_cl[Cellbegin];

#define INTER(triTable, edgeTable) \
    const unsigned int edge = triTable[m_data.m_caseNums[ValidCellIndex]][idx]; \
    const unsigned int v1 = edgeTable[0][edge]; \
    const unsigned int v2 = edgeTable[1][edge]; \
    const Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]); \
    Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx; \
    for(int j = 0; j < m_data.m_numInVertData; j++) { \
        m_data.m_outVertPtr[j][outvertexindex] = \
            lerp(m_data.m_inVertPtr[j][cl[v1]], m_data.m_inVertPtr[j][cl[v2]], t); \
    } \
    for(int j = 0; j < m_data.m_numInVertDataI; j++) { \
        m_data.m_outVertPtrI[j][outvertexindex] = \
            lerp(m_data.m_inVertPtrI[j][cl[v1]], m_data.m_inVertPtrI[j][cl[v2]], t); \
    }

      for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]/3; idx++) {
          Index outcellindex = m_data.m_LocationList[ValidCellIndex]/3+idx; \
          for(int j = 0; j < m_data.m_numInCellData; j++) {
              m_data.m_outCellPtr[j][outcellindex] = m_data.m_inCellPtr[j][CellNr];
          }
          for(int j = 0; j < m_data.m_numInCellDataI; j++) {
              m_data.m_outCellPtrI[j][outcellindex] = m_data.m_inCellPtrI[j][CellNr];
          }
      }

      switch (m_data.m_tl[CellNr] & ~UnstructuredGrid::CONVEX_BIT) {

         case UnstructuredGrid::HEXAHEDRON: {

            Scalar field[8];
            for (int idx = 0; idx < 8; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(hexaTriTable, hexaEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::TETRAHEDRON: {

            Scalar field[4];
            for (int idx = 0; idx < 4; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(tetraTriTable, tetraEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PYRAMID: {

            Scalar field[5];
            for (int idx = 0; idx < 5; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(pyrTriTable, pyrEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PRISM: {

            Scalar field[6];
            for (int idx = 0; idx < 6; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(prismTriTable, prismEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::POLYHEDRON: {
            /* find all iso-points on each edge of each face,
               build a triangle for each consecutive pair and a center point,
               orient outwards towards smaller values */

            const auto &cl = m_data.m_cl;

            Index numAvg = 0;
            Scalar middleData[MaxNumData];
            Index middleDataI[MaxNumData];
            for(int i = 0; i < MaxNumData; i++ ){
               middleData[i] = 0;
               middleDataI[i] = 0;
            };
            Scalar cd1[MaxNumData], cd2[MaxNumData];
            Index cd1I[MaxNumData], cd2I[MaxNumData];

            Index outIdx = m_data.m_LocationList[ValidCellIndex];
            for (Index i = Cellbegin; i < Cellend; i += cl[i]+1) {

               const Index nvert = cl[i];
               Index c1 = cl[i+nvert];
               bool flipped = false, haveIsect = false;
               for (Index k=i+1; k<i+nvert+1; ++k) {
                   const Index c2 = cl[k];

                   for(int i = 0; i < m_data.m_numInVertData; i++){
                       cd1[i] = m_data.m_inVertPtr[i][c1];
                       cd2[i] = m_data.m_inVertPtr[i][c2];
                   }
                   for(int i = 0; i < m_data.m_numInVertDataI; i++){
                       cd1I[i] = m_data.m_inVertPtrI[i][c1];
                       cd2I[i] = m_data.m_inVertPtrI[i][c2];
                   }

                   Scalar d1 = m_data.m_isoFunc(c1);
                   Scalar d2 = m_data.m_isoFunc(c2);

                   bool smallToBig = d1 <= m_data.m_isovalue && d2 > m_data.m_isovalue;
                   bool bigToSmall = d1 > m_data.m_isovalue && d2 <= m_data.m_isovalue;

                   if (smallToBig || bigToSmall) {
                       if (!haveIsect) {
                           flipped = bigToSmall;
                           haveIsect = true;
                       }
                       Index out = outIdx;
                       if (flipped) {
                           if (bigToSmall)
                               out += 1;
                           else
                               out -= 1;
                       }
                       Scalar t = tinterp(m_data.m_isovalue, d1, d2);
                       for(int i = 0; i < m_data.m_numInVertData; i++) {
                           Scalar v = lerp(cd1[i], cd2[i], t);
                           middleData[i] += v;
                           m_data.m_outVertPtr[i][out] = v;
                       }
                       for(int i = 0; i < m_data.m_numInVertDataI; i++){
                           Index vI = lerp(cd1I[i], cd2I[i], t);
                           middleDataI[i] += vI;
                           m_data.m_outVertPtrI[i][out] = vI;
                       }

                       ++outIdx;
                       if (bigToSmall^flipped)
                           ++outIdx;
                       ++numAvg;
                   }

                   c1 = c2;
               }
            }
            if (numAvg > 0) {
                for(int i = 0; i < m_data.m_numInVertData; i++){
                    middleData[i] /= numAvg;
                }
                for(int i = 0; i < m_data.m_numInVertDataI; i++){
                    middleDataI[i] /= numAvg;
                }
            }
            for (Index i = 2; i < numVert; i += 3) {
               const Index idx = m_data.m_LocationList[ValidCellIndex]+i;
               for(int i = 0; i < m_data.m_numInVertData; i++){
                  m_data.m_outVertPtr[i][idx] = middleData[i];
               }
               for(int i = 0; i < m_data.m_numInVertDataI; i++){
                  m_data.m_outVertPtrI[i][idx] = middleDataI[i];
               }
            };
            break;
         }
      }
   }
};

template<class Data>
struct checkcell {

   typedef float argument_type;
   typedef float result_type;
   Data &m_data;
   checkcell(Data &data) : m_data(data) {}

   __host__ __device__ int operator()(const thrust::tuple<Index,Index> iCell) const {

      int havelower = 0;
      int havehigher = 0;
      Index Cell = iCell.get<0>();
      Index nextCell = iCell.get<1>();
      for (Index i=Cell; i<nextCell; i++) {
         float val = m_data.m_isoFunc(m_data.m_cl[i]);
         if (val>m_data.m_isovalue) {
            havelower=1;
            if (havehigher)
               return 1;
         } else {
            havehigher=1;
            if (havelower)
               return 1;
         }
      }
      return 0;
   }
};

template<class Data>
struct classify_cell {

   classify_cell(Data &data) : m_data(data) {}

   Data &m_data;

   __host__ __device__ thrust::tuple<Index,Index> operator()(Index CellNr) {

      const auto &cl = m_data.m_cl;

      int tableIndex = 0;
      Index begin = m_data.m_el[CellNr], end = m_data.m_el[CellNr+1];
      Index nvert = end-begin;
      unsigned char CellType = m_data.m_tl[CellNr] & ~UnstructuredGrid::CONVEX_BIT;
      int numVerts = 0;
      if (CellType != UnstructuredGrid::POLYHEDRON) {
         for (Index idx = 0; idx < nvert; idx ++) {
            tableIndex += (((int) (m_data.m_isoFunc(m_data.m_cl[begin+idx]) > m_data.m_isovalue)) << idx);
         }
      }
      switch (CellType) {

         case UnstructuredGrid::HEXAHEDRON:
            numVerts = hexaNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::TETRAHEDRON:
            numVerts = tetraNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PYRAMID:
            numVerts = pyrNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PRISM:
            numVerts = prismNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::POLYHEDRON: {

            Index vertcounter = 0;
            for (Index i = begin; i < end; i += cl[i]+1) {
               const Index N = cl[i];
               Index prev = cl[i+N];
               for (Index k=i+1; k<i+N+1; ++k) {
                   Index v = cl[k];

                   if (m_data.m_isoFunc(prev) <= m_data.m_isovalue && m_data.m_isoFunc(v) > m_data.m_isovalue) {
                       ++vertcounter;
                   } else if(m_data.m_isoFunc(prev) > m_data.m_isovalue && m_data.m_isoFunc(v) <= m_data.m_isovalue) {
                       ++vertcounter;
                   }

                   prev = v;
               }
            }
            numVerts = vertcounter + vertcounter/2;
            break;
         }
      };
      return thrust::make_tuple<Index, Index> (tableIndex, numVerts);
   }
};

Leveller::Leveller(const IsoController &isocontrol, UnstructuredGrid::const_ptr grid, const Scalar isovalue, Index processortype)
      : m_isocontrol(isocontrol)
      , m_grid(grid)
      , m_isoValue(isovalue)
      , m_processortype(processortype)
      , gmin(std::numeric_limits<Scalar>::max())
      , gmax(-std::numeric_limits<Scalar>::max())
      , m_objectTransform(grid->getTransform())
   {
      m_triangles = Triangles::ptr(new Triangles(Object::Initialized));
      m_triangles->setMeta(grid->meta());
   }

template<class Data, class pol>
Index Leveller::calculateSurface(Data &data) {

   thrust::counting_iterator<int> first(0);
   thrust::counting_iterator<int> last = first + m_grid->getNumElements();
   typedef thrust::tuple<typename Data::IndexIterator, typename Data::IndexIterator> Iteratortuple;
   typedef thrust::zip_iterator<Iteratortuple> ZipIterator;
   ZipIterator ElTupleVec(thrust::make_tuple(&data.m_el[0], &data.m_el[1]));
   data.m_ValidCellVector.resize(m_grid->getNumElements());
   typename Data::VectorIndexIterator end = thrust::copy_if(pol(), first, last, ElTupleVec, data.m_ValidCellVector.begin(), checkcell<Data>(data));
   size_t numValidCells = end-data.m_ValidCellVector.begin();
   data.m_caseNums.resize(numValidCells);
   data.m_numVertices.resize(numValidCells);
   data.m_LocationList.resize(numValidCells);
   thrust::transform(pol(), data.m_ValidCellVector.begin(), end, thrust::make_zip_iterator(thrust::make_tuple(data.m_caseNums.begin(),data.m_numVertices.begin())), classify_cell<Data>(data));
   thrust::exclusive_scan(pol(), data.m_numVertices.begin(), data.m_numVertices.end(), data.m_LocationList.begin());
   Index totalNumVertices = 0;
   if (!data.m_numVertices.empty())
      totalNumVertices += data.m_numVertices.back();
   if (!data.m_LocationList.empty())
      totalNumVertices += data.m_LocationList.back();
   for(int i = 0; i < data.m_numInVertData; i++){
      data.m_outVertData[i]->resize(totalNumVertices);
   }
   for(int i = 0; i < data.m_numInVertDataI; i++){
      data.m_outVertDataI[i]->resize(totalNumVertices);
   }
   for (int i=0; i<data.m_numInCellData; ++i) {
       data.m_outCellData[i]->resize(totalNumVertices/3);
   }
   for (int i=0; i<data.m_numInCellDataI; ++i) {
       data.m_outCellDataI[i]->resize(totalNumVertices/3);
   }
   thrust::counting_iterator<Index> start(0), finish(numValidCells);
   thrust::for_each(pol(), start, finish, process_Cell<Data>(data));
   return totalNumVertices;
}

bool Leveller::process() {
#ifndef CUTTINGSURFACE
   Vec<Scalar>::const_ptr dataobj = Vec<Scalar>::as(m_data);
   if (!dataobj)
      return false;
#else
#endif

   Index totalNumVertices = 0;

   switch (m_processortype) {

      case Host: {

         HostData HD(m_isoValue,
#ifndef CUTTINGSURFACE
               m_isocontrol.newFunc(m_grid->getTransform(), &dataobj->x()[0]),
#else
               m_isocontrol.newFunc(m_grid->getTransform(), &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0]),
#endif
               m_grid->el(), m_grid->tl(), m_grid->cl(), m_grid->x(), m_grid->y(), m_grid->z());

         for (size_t i=0; i<m_vertexdata.size(); ++i) {
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_vertexdata[i])){
               HD.addmappeddata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_vertexdata[i])){
               HD.addmappeddata(Vect->x());
               HD.addmappeddata(Vect->y());
               HD.addmappeddata(Vect->z());
            }
            if(Vec<Index,1>::const_ptr Idx = Vec<Index,1>::as(m_vertexdata[i])){
               HD.addmappeddata(Idx->x());
            }
         }
         for (size_t i=0; i<m_celldata.size(); ++i) {
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_celldata[i])){
               HD.addcelldata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_celldata[i])){
               HD.addcelldata(Vect->x());
               HD.addcelldata(Vect->y());
               HD.addcelldata(Vect->z());
            }
            if(Vec<Index,1>::const_ptr Idx = Vec<Index,1>::as(m_celldata[i])){
               HD.addcelldata(Idx->x());
            }
         }

         totalNumVertices = calculateSurface<HostData, thrust::detail::host_t>(HD);

         {
             size_t idx=0;
             m_triangles->d()->x[0] = HD.m_outVertData[idx++];
             m_triangles->d()->x[1] = HD.m_outVertData[idx++];
             m_triangles->d()->x[2] = HD.m_outVertData[idx++];

             size_t idxI=0;
             for (size_t i=0; i<m_vertexdata.size(); ++i) {
                 if(Vec<Scalar>::as(m_vertexdata[i])){

                     Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertData[idx++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);

                 }
                 if(Vec<Scalar,3>::as(m_vertexdata[i])){

                     Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertData[idx++];
                     out->d()->x[1] = HD.m_outVertData[idx++];
                     out->d()->x[2] = HD.m_outVertData[idx++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);

                 }
                 if(Vec<Index>::as(m_vertexdata[i])){

                     Vec<Index>::ptr out = Vec<Index>::ptr(new Vec<Index>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertDataI[idxI++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);
                 }
             }
         }
         {
             size_t idx=0;
             size_t idxI=0;
             for (size_t i=0; i<m_celldata.size(); ++i) {
                 if(Vec<Scalar>::as(m_celldata[i])){

                     Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellData[idx++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);

                 }
                 if(Vec<Scalar,3>::as(m_celldata[i])){

                     Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellData[idx++];
                     out->d()->x[1] = HD.m_outCellData[idx++];
                     out->d()->x[2] = HD.m_outCellData[idx++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);

                 }
                 if(Vec<Index>::as(m_celldata[i])){

                     Vec<Index>::ptr out = Vec<Index>::ptr(new Vec<Index>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellDataI[idxI++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);
                 }
             }
         }
         break;
      }

      case Device: {

         DeviceData DD(m_isoValue,
#ifndef CUTTINGSURFACE
               m_isocontrol.newFunc(m_grid->getTransform(), &dataobj->x()[0]),
#else
               m_isocontrol.newFunc(m_grid->getTransform(), &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0]),
#endif
               m_grid->getNumElements(), m_grid->el(), m_grid->tl(), m_grid->getNumCorners(), m_grid->cl(), m_grid->getSize(), m_grid->x(), m_grid->y(), m_grid->z());

#if 0
         totalNumVertices = calculateSurface<DeviceData, thrust::device>(DD);
#endif

         m_triangles->x().resize(totalNumVertices);
         Scalar *out_x = m_triangles->x().data();
         thrust::copy(DD.m_outVertData[0]->begin(), DD.m_outVertData[0]->end(), out_x);

         m_triangles->y().resize(totalNumVertices);
         Scalar *out_y = m_triangles->y().data();
         thrust::copy(DD.m_outVertData[1]->begin(), DD.m_outVertData[1]->end(), out_y);

         m_triangles->z().resize(totalNumVertices);
         Scalar *out_z = m_triangles->z().data();
         thrust::copy(DD.m_outVertData[2]->begin(), DD.m_outVertData[2]->end(), out_z);

         if(m_vertexdata.size()){
            if(Vec<Scalar>::as(m_vertexdata[0])){

               Vec<Scalar>::ptr out = Vec<Scalar>::ptr(new Vec<Scalar>(totalNumVertices));
               thrust::copy(DD.m_outVertData[3]->begin(), DD.m_outVertData[3]->end(), out->x().data());
               out->setMeta(m_vertexdata[0]->meta());
               m_outvertData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_vertexdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(totalNumVertices));
               thrust::copy(DD.m_outVertData[3]->begin(), DD.m_outVertData[3]->end(), out->x().data());
               thrust::copy(DD.m_outVertData[4]->begin(), DD.m_outVertData[4]->end(), out->y().data());
               thrust::copy(DD.m_outVertData[5]->begin(), DD.m_outVertData[5]->end(), out->z().data());
               out->setMeta(m_vertexdata[0]->meta());
               m_outvertData.push_back(out);

            }
         }
         break;
      }
   }

   return true;
}

#ifndef CUTTINGSURFACE
void Leveller::setIsoData(Vec<Scalar>::const_ptr obj) {
   m_data = obj;
}
#endif

void Leveller::addMappedData(DataBase::const_ptr mapobj ){
    if (mapobj->mapping() == DataBase::Element)
        m_celldata.push_back(mapobj);
    else
        m_vertexdata.push_back(mapobj);
}

Object::ptr Leveller::result() {
      return m_triangles;
   }

DataBase::ptr Leveller::mapresult() const {
   if(m_outvertData.size())
      return m_outvertData[0];
   else if(m_outcellData.size())
      return m_outcellData[0];
   else
      return DataBase::ptr();
}

DataBase::ptr Leveller::cellresult() const {
   if(m_outcellData.size())
      return m_outcellData[0];
   else
      return DataBase::ptr();
}

std::pair<Scalar, Scalar> Leveller::range() {
   return std::make_pair(gmin, gmax);
}
